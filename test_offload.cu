#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel to add two arrays
__global__ void add_arrays(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 1 << 20; // Array size (1 million elements)
    const size_t bytes = N * sizeof(float);

    // Allocate host memory
    float *h_a = new float[N];
    float *h_b = new float[N];
    float *h_c = new float[N];

    // Initialize input arrays
    for (int i = 0; i < N; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Launch kernel (1024 threads per block, enough blocks to cover N elements)
    int threads_per_block = 1024;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
    add_arrays<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Verify result
    bool success = true;
    for (int i = 0; i < N; ++i) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            success = false;
            break;
        }
    }

    if (success) {
        std::cout << "Test PASSED!" << std::endl;
    } else {
        std::cout << "Test FAILED!" << std::endl;
    }

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
